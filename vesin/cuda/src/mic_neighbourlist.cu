#include "hip/hip_runtime.h"
#include "mic_neighbourlist.cuh"
#include <hip/hip_runtime.h>

#define NWARPS 4
#define WARP_SIZE 32

#ifndef MAX_NEIGHBOURS_PER_ATOM
#define MAX_NEIGHBOURS_PER_ATOM 1024 // Make configurable
#endif

__device__ inline long atomicAdd(long *address, long val) {
  unsigned long long *address_as_ull =
      reinterpret_cast<unsigned long long *>(address);
  unsigned long long old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(
        address_as_ull, assumed,
        static_cast<unsigned long long>(val + static_cast<long>(assumed)));
  } while (assumed != old);

  return static_cast<long>(old);
}

__device__ inline unsigned long atomicAdd(unsigned long *address,
                                          unsigned long val) {
  unsigned long long *address_as_ull =
      reinterpret_cast<unsigned long long *>(address);
  unsigned long long old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    static_cast<unsigned long long>(
                        val + static_cast<unsigned long>(assumed)));
  } while (assumed != old);

  return static_cast<unsigned long>(old);
}

// ops for vector type deduction
__device__ inline float3 operator-(const float3 &a, const float3 &b) {
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ inline double3 operator-(const double3 &a, const double3 &b) {
  return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ inline float dot(const float3 &a, const float3 &b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ inline double dot(const double3 &a, const double3 &b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

// Vector3IO template structure for handling vectorized types
template <typename scalar_t> struct Vector3IO;

/* template structure for dealing with float3, double3 vectorized types */
template <> struct Vector3IO<float> {
  using scalar_t = float;
  using vec_t = float3;

  __device__ static void unpack(const vec_t &v, scalar_t &x0, scalar_t &x1,
                                scalar_t &x2) {
    x0 = v.x;
    x1 = v.y;
    x2 = v.z;
  }

  __device__ static vec_t pack(scalar_t x0, scalar_t x1, scalar_t x2) {
    return {x0, x1, x2};
  }
};

template <> struct Vector3IO<double> {
  using scalar_t = double;
  using vec_t = double3;

  __device__ static void unpack(const vec_t &v, scalar_t &x0, scalar_t &x1,
                                scalar_t &x2) {
    x0 = v.x;
    x1 = v.y;
    x2 = v.z;
  }

  __device__ static vec_t pack(scalar_t x0, scalar_t x1, scalar_t x2) {
    return {x0, x1, x2};
  }
};

template <typename scalar_t>
__device__ typename Vector3IO<scalar_t>::vec_t
operator+(const typename Vector3IO<scalar_t>::vec_t &a,
          const typename Vector3IO<scalar_t>::vec_t &b) {
  return Vector3IO<scalar_t>::pack(a.x + b.x, a.y + b.y, a.z + b.z);
}

template <typename scalar_t>
__device__ typename Vector3IO<scalar_t>::vec_t
operator-(const typename Vector3IO<scalar_t>::vec_t &a,
          const typename Vector3IO<scalar_t>::vec_t &b) {
  return Vector3IO<scalar_t>::pack(a.x - b.x, a.y - b.y, a.z - b.z);
}

template <typename scalar_t>
__device__ scalar_t dot(const typename Vector3IO<scalar_t>::vec_t &a,
                        const typename Vector3IO<scalar_t>::vec_t &b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

template <typename scalar_t>
__device__ void invert_cell_matrix(const scalar_t *cell, scalar_t *inv_cell) {
  scalar_t a = cell[0], b = cell[1], c = cell[2];
  scalar_t d = cell[3], e = cell[4], f = cell[5];
  scalar_t g = cell[6], h = cell[7], i = cell[8];

  scalar_t det =
      a * (e * i - f * h) - b * (d * i - f * g) + c * (d * h - e * g);
  scalar_t invdet = scalar_t(1.0) / det;

  inv_cell[0] = (e * i - f * h) * invdet;
  inv_cell[1] = (c * h - b * i) * invdet;
  inv_cell[2] = (b * f - c * e) * invdet;
  inv_cell[3] = (f * g - d * i) * invdet;
  inv_cell[4] = (a * i - c * g) * invdet;
  inv_cell[5] = (c * d - a * f) * invdet;
  inv_cell[6] = (d * h - e * g) * invdet;
  inv_cell[7] = (b * g - a * h) * invdet;
  inv_cell[8] = (a * e - b * d) * invdet;
}
template <typename scalar_t>
__device__ void
apply_periodic_boundary(typename Vector3IO<scalar_t>::vec_t &displacement,
                        int3 &shift, const scalar_t *cell,
                        const scalar_t *inv_cell) {
  using vec_t = typename Vector3IO<scalar_t>::vec_t;

  // 1) project into fractional coords
  vec_t frac;
  frac.x = displacement.x * inv_cell[0] + displacement.y * inv_cell[1] +
           displacement.z * inv_cell[2];
  frac.y = displacement.x * inv_cell[3] + displacement.y * inv_cell[4] +
           displacement.z * inv_cell[5];
  frac.z = displacement.x * inv_cell[6] + displacement.y * inv_cell[7] +
           displacement.z * inv_cell[8];

  // 2) determine how many whole cells we cross in each direction
  int sx = static_cast<int>(round(frac.x));
  int sy = static_cast<int>(round(frac.y));
  int sz = static_cast<int>(round(frac.z));

  shift.x = sx;
  shift.y = sy;
  shift.z = sz;

  // 3) wrap fractional back into [-0.5,0.5] by subtracting the integer part
  frac.x -= sx;
  frac.y -= sy;
  frac.z -= sz;

  // 4) reconstruct the Cartesian displacement inside the primary cell
  vec_t wrapped;
  wrapped.x = frac.x * cell[0] + frac.y * cell[3] + frac.z * cell[6];
  wrapped.y = frac.x * cell[1] + frac.y * cell[4] + frac.z * cell[7];
  wrapped.z = frac.x * cell[2] + frac.y * cell[5] + frac.z * cell[8];

  // 5) overwrite the input
  displacement = wrapped;
}

template <typename scalar_t>
__global__ void compute_mic_neighbours_full_impl(
    const scalar_t *__restrict__ positions, const scalar_t *cell, long nnodes,
    scalar_t cutoff, unsigned long *pair_counter,
    unsigned long *__restrict__ edge_indices, int *__restrict__ shifts,
    scalar_t *__restrict__ distances, scalar_t *__restrict__ vectors,
    bool return_shifts, bool return_distances, bool return_vectors, bool full) {

  using vec_t = typename Vector3IO<scalar_t>::vec_t;

  __shared__ unsigned long edge_pair_shared[NWARPS];
  __shared__ long edge_indices_shared[MAX_NEIGHBOURS_PER_ATOM * NWARPS];
  __shared__ scalar_t inv_cell[9];

  const int warp_id = threadIdx.y;
  const int thread_in_warp = threadIdx.x;
  const int node_index = blockIdx.x * blockDim.y + warp_id;
  const scalar_t cutoff2 = cutoff * cutoff;

  if (thread_in_warp == 0)
    edge_pair_shared[warp_id] = 0;

  if (cell != nullptr && thread_in_warp == 0 && warp_id == 0)
    invert_cell_matrix(cell, inv_cell);

  __syncthreads(); // Ensure inv_cell is ready

  if (node_index >= nnodes)
    return;

  vec_t ri = *reinterpret_cast<const vec_t *>(&positions[node_index * 3]);

  for (int j = thread_in_warp; j < nnodes; j += blockDim.x) {
    vec_t rj = *reinterpret_cast<const vec_t *>(&positions[j * 3]);

    vec_t disp = ri - rj;
    int3 shift = make_int3(0, 0, 0); // Initialize shift
    if (cell != nullptr)
      apply_periodic_boundary<scalar_t>(disp, shift, cell, inv_cell);

    scalar_t dist2 = dot(disp, disp);

    if (dist2 < cutoff2 && dist2 > scalar_t(0.0)) {
      int edge_local = atomicAdd(&edge_pair_shared[warp_id], 1);
      if (edge_local < MAX_NEIGHBOURS_PER_ATOM) {
        edge_indices_shared[warp_id * MAX_NEIGHBOURS_PER_ATOM + edge_local] = j;
      }
    }
  }

  __syncwarp();

  int iglobal = 0;
  if (thread_in_warp == 0) {
    iglobal = atomicAdd(pair_counter, (long)edge_pair_shared[warp_id]);
  }

  iglobal = __shfl_sync(0xFFFFFFFF, iglobal, 0); // Broadcast iglobal

  int num_edges = edge_pair_shared[warp_id];
  for (int j = thread_in_warp; j < num_edges; j += WARP_SIZE) {
    int edge_idx = edge_indices_shared[warp_id * MAX_NEIGHBOURS_PER_ATOM + j];

    edge_indices[iglobal + j] = node_index; // receiver
    edge_indices[(nnodes * MAX_NEIGHBOURS_PER_ATOM) + iglobal + j] =
        edge_idx; // sender

    vec_t rj = *reinterpret_cast<const vec_t *>(&positions[edge_idx * 3]);
    vec_t disp = ri - rj;

    scalar_t dist2 = dot(disp, disp);
    int3 shift = make_int3(0, 0, 0);

    if (cell != nullptr)
      apply_periodic_boundary<scalar_t>(disp, shift, cell, inv_cell);

    if (return_shifts) {
      reinterpret_cast<int3 &>(shifts[(iglobal + j) * 3]) = shift;
    }

    if (return_vectors) {
      reinterpret_cast<vec_t &>(vectors[(iglobal + j) * 3]) = disp;
    }

    if (return_distances) {
      distances[iglobal + j] = sqrt(dist2);
    }
  }
}

template <typename scalar_t>
void vesin::cuda::compute_mic_neighbourlist(
    const scalar_t *positions, const scalar_t *cell, long nnodes,
    scalar_t cutoff, unsigned long *pair_counter, unsigned long *edge_indices,
    int *shifts, scalar_t *distances, scalar_t *vectors, bool return_shifts,
    bool return_distances, bool return_vectors, bool full) {

  dim3 blockDim(WARP_SIZE, NWARPS);
  dim3 gridDim((nnodes + NWARPS - 1) / NWARPS);

  compute_mic_neighbours_full_impl<scalar_t><<<gridDim, blockDim>>>(
      positions, cell, nnodes, cutoff, pair_counter, edge_indices, shifts,
      distances, vectors, // pass them through
      return_shifts, return_distances, return_vectors, full);
}

// Explicit instantiation for double
template void vesin::cuda::compute_mic_neighbourlist<double>(
    const double *positions, const double *cell, long nnodes, double cutoff,
    unsigned long *pair_counter, unsigned long *edge_indices, int *shifts,
    double *distances, double *vectors, bool return_shifts,
    bool return_distances, bool return_vectors, bool full);

// Explicit instantiation for float
template void vesin::cuda::compute_mic_neighbourlist<float>(
    const float *positions, const float *cell, long nnodes, float cutoff,
    unsigned long *pair_counter, unsigned long *edge_indices, int *shifts,
    float *distances, float *vectors, bool return_shifts, bool return_distances,
    bool return_vectors, bool full);